#include "hip/hip_runtime.h"
#include "rawspec.h"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>

#define NO_PLAN   ((hipfftHandle)-1)
#define NO_STREAM ((hipStream_t)-1)

#define PRINT_ERRMSG(error)                  \
  fprintf(stderr, "got error %s at %s:%d\n", \
      _cudaGetErrorEnum(error),  \
      __FILE__, __LINE__)

// Stream callback data structure
typedef struct {
  rawspec_context * ctx;
  int output_product;
} dump_cb_data_t;

// GPU context structure
typedef struct {
  // Device pointer to FFT input buffer
  char2 * d_fft_in;
  // Array of device pointers to FFT output buffers
  hipfftComplex * d_fft_out[MAX_OUTPUTS];
  // Array of device pointers to power buffers
  float * d_pwr_out[MAX_OUTPUTS];
  // Array of handles to FFT plans
  hipfftHandle plan[MAX_OUTPUTS];
  // Array of Ns values (number of specta (FFTs) per input buffer for Nt)
  unsigned int Nss[MAX_OUTPUTS];
  // Array of hipStream_t values
  hipStream_t stream[MAX_OUTPUTS];
  // Array of grids for accumulate kernel
  dim3 grid[MAX_OUTPUTS];
  // Array of number of threads to use per block for accumulate kernel
  int nthreads[MAX_OUTPUTS];
  // Array of Nd values (number of spectra per dump)
  unsigned int Nds[MAX_OUTPUTS];
  // Array of Ni values (number of input buffers per dump)
  unsigned int Nis[MAX_OUTPUTS];
  // A count of the number of input buffers processed
  unsigned int inbuf_count;
  // Array of dump_cb_data_t structures for dump callback
  dump_cb_data_t dump_cb_data[MAX_OUTPUTS];
  // Flag indicating that the caller is managing the input block buffers
  // Non-zero when caller is managing (i.e. allocating and freeing) the
  // buffers; zero when we are.
  int caller_managed;
} rawspec_gpu_context;

// Texture declarations
texture<char, 2, hipReadModeNormalizedFloat> char_tex;

__device__ hipfftComplex load_callback(void *p_v_in,
                                      size_t offset,
                                      void *p_v_user,
                                      void *p_v_shared)
{
  hipfftComplex c;
  offset += (hipfftComplex *)p_v_in - (hipfftComplex *)p_v_user;
  c.x = tex2D(char_tex, ((2*offset  ) & 0x7fff), ((  offset  ) >> 14));
  c.y = tex2D(char_tex, ((2*offset+1) & 0x7fff), ((2*offset+1) >> 15));
  return c;
}

__device__ void store_callback(void *p_v_out,
                               size_t offset,
                               hipfftComplex element,
                               void *p_v_user,
                               void *p_v_shared)
{
  float pwr = element.x * element.x + element.y * element.y;
  ((float *)p_v_user)[offset] += pwr;
}

__device__ hipfftCallbackLoadC d_cufft_load_callback = load_callback;
__device__ hipfftCallbackStoreC d_cufft_store_callback = store_callback;

#define MAX_THREADS (1024)

// Accumulate kernel
__global__ void accumulate(float * pwr_buf, unsigned int Na, size_t xpitch, size_t ypitch, size_t zpitch)
{
  unsigned int i;

  // TODO Add check for past end of spectrum

  off_t offset0 = blockIdx.z * zpitch
                + blockIdx.y * ypitch
                + blockIdx.x * MAX_THREADS
                + threadIdx.x;

  off_t offset = offset0;

  float sum = pwr_buf[offset];

  for(i=1; i<Na; i++) {
    offset += xpitch;
    sum += pwr_buf[offset];
  }

  pwr_buf[offset0] = sum;
}

// Stream callback function that is called right after an output product's GPU
// power buffer has been copied to the host power buffer.
static void CUDART_CB dump_stream_callback(hipStream_t stream,
                                           hipError_t status,
                                           void *data)
{
  dump_cb_data_t * dump_cb_data = (dump_cb_data_t *)data;
  if(dump_cb_data->ctx->dump_callback) {
    dump_cb_data->ctx->dump_callback(dump_cb_data->ctx,
                                     dump_cb_data->output_product);
  }
}

// Sets ctx->Ntmax.
// Allocates host and device buffers based on the ctx->N values.
// Allocates and sets the ctx->gpu_ctx field.
// Creates CuFFT plans.
// Creates streams.
// Returns 0 on success, non-zero on error.
int rawspec_initialize(rawspec_context * ctx)
{
  int i;
  size_t inbuf_size;
  hipError_t cuda_rc;
  hipfftResult cufft_rc;

  // Host copies of cufft callback pointers
  hipfftCallbackLoadC h_cufft_load_callback;
  hipfftCallbackStoreC h_cufft_store_callback;

  // Validate No
  if(ctx->No == 0 || ctx->No > MAX_OUTPUTS) {
    fprintf(stderr, "output products must be in range [1..%d], not %d\n",
        MAX_OUTPUTS, ctx->No);
    return 1;
  }

  // Validate Np
  if(ctx->Np == 0 || ctx->Np > 2) {
    fprintf(stderr,
        "number of polarizations must be in range [1..2], not %d\n", ctx->Np);
    return 1;
  }

  // Validate Ntpb
  if(ctx->Ntpb == 0) {
    fprintf(stderr, "number of time samples per block cannot be zero\n");
    return 1;
  }

  // Determine Ntmax (and validate Nts)
  ctx->Ntmax = 0;
  for(i=0; i<ctx->No; i++) {
    if(ctx->Nts[i] == 0) {
      fprintf(stderr, "Nts[%d] cannot be 0\n", i);
      return 1;
    }
    if(ctx->Ntmax < ctx->Nts[i]) {
      ctx->Ntmax = ctx->Nts[i];
    }
  }
  // Validate that all Nts are factors of Ntmax.  This constraint helps
  // simplify input buffer management.
  for(i=0; i<ctx->No; i++) {
    if(ctx->Ntmax % ctx->Nts[i] != 0) {
      fprintf(stderr, "Nts[%d] (%u) is not a factor of Ntmax (%u)\n",
          i, ctx->Nts[i], ctx->Ntmax);
      return 1;
    }
  }

  // Validate/calculate Nb
  // If ctx->Nb is given by caller (i.e. is non-zero)
  if(ctx->Nb != 0) {
    // Validate that Ntmax is a factor of (Nb * Ntpb)
    if((ctx->Nb * ctx->Ntpb) % ctx->Ntmax != 0) {
      fprintf(stderr,
          "Ntmax (%u) is not a factor of Nb*Ntpb (%u * %u = %u)\n",
          ctx->Ntmax, ctx->Nb, ctx->Ntpb, ctx->Nb*ctx->Ntpb);
      return 1;
    }
  } else {
    // Cannot calculate Nb for caller-managed h_blkbufs
    if(ctx->h_blkbufs) {
      fprintf(stderr,
          "Must specify number of input blocks when caller-managed\n");
      return 1;
    }

    // Calculate Nb
    // If Ntmax is less than one block
    if(ctx->Ntmax < ctx->Ntpb) {
      // Validate that Ntmax is a factor of Ntpb
      if(ctx->Ntpb % ctx->Ntmax != 0) {
        fprintf(stderr, "Ntmax (%u) is not a factor of Ntpb (%u)\n",
            ctx->Ntmax, ctx->Ntpb);
        return 1;
      }
      ctx->Nb = 1;
    } else {
      // Validate that Ntpb is factor of Ntmax
      if(ctx->Ntmax % ctx->Ntpb != 0) {
        fprintf(stderr, "Ntpb (%u) is not a factor of Nmax (%u)\n",
            ctx->Ntpb, ctx->Ntmax);
        return 1;
      }
      ctx->Nb = ctx->Ntmax / ctx->Ntpb;
    }
  }

  // Validate Nas
  for(i=0; i < ctx->No; i++) {
    if(ctx->Nas[i] == 0) {
      fprintf(stderr, "Nas[%d] cannot be 0\n", i);
      return 1;
    }
    // If mulitple integrations per input buffer
    if(ctx->Nts[i]*ctx->Nas[i] < ctx->Nb*ctx->Ntpb) {
      // Must have integer integrations per input buffer
      if((ctx->Nb * ctx->Ntpb) % (ctx->Nts[i] * ctx->Nas[i]) != 0) {
        fprintf(stderr,
            "Nts[%d] * Nas[%d] (%u * %u) must divide Nb * Ntpb (%u * %u)\n",
            i, i, ctx->Nts[i], ctx->Nas[i], ctx->Nb, ctx->Ntpb);
        return 1;
      }
    } else {
      // Must have integer input buffers per integration
      if((ctx->Nts[i] * ctx->Nas[i]) % (ctx->Nb * ctx->Ntpb) != 0) {
        fprintf(stderr,
            "Nb * Ntpb (%u * %u) must divide Nts[%d] * Nas[%d] (%u * %u)\n",
            ctx->Nb, ctx->Ntpb, i, i, ctx->Nts[i], ctx->Nas[i]);
        return 1;
      }
    }
  }

  // Null out all pointers
  // TODO Add support for client managed host buffers
  for(i=0; i < MAX_OUTPUTS; i++) {
    ctx->h_pwrbuf[i] = NULL;
  }
  ctx->gpu_ctx = NULL;

  // Allocate GPU context
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)malloc(sizeof(rawspec_gpu_context));

  if(!gpu_ctx) {
    rawspec_cleanup(ctx);
    return 1;
  }

  // Store pointer to gpu_ctx in ctx
  ctx->gpu_ctx = gpu_ctx;

  // NULL out pointers (and invalidate plans)
  gpu_ctx->d_fft_in = NULL;
  for(i=0; i<MAX_OUTPUTS; i++) {
    gpu_ctx->d_fft_out[i] = NULL;
    gpu_ctx->d_pwr_out[i] = NULL;
    gpu_ctx->plan[i] = NO_PLAN;
    gpu_ctx->stream[i] = NO_STREAM;
    gpu_ctx->dump_cb_data[i].ctx = ctx;
    gpu_ctx->dump_cb_data[i].output_product = i;
  }

  // Initialize inbuf_count
  gpu_ctx->inbuf_count = 0;

  if(!ctx->h_blkbufs) {
    // Remember that we (not the caller) are managing these buffers
    // (i.e. we will need to free them when cleaning up).
    gpu_ctx->caller_managed = 0;

    // Alllocate host input block buffers
    ctx->h_blkbufs = (char **)malloc(ctx->Nb * sizeof(char *));
    for(i=0; i < ctx->Nb; i++) {
      // Block buffer can use write combining
      cuda_rc = hipHostAlloc(&ctx->h_blkbufs[i],
                         ctx->Ntpb*ctx->Np*ctx->Nc*sizeof(char2),
                         hipHostMallocWriteCombined);
      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        return 1;
      }
    }
  } else {
    // Remember that the caller is managing these buffers
    // (i.e. we will only need to unregister them when cleaning up).
    gpu_ctx->caller_managed = 1;

    // Register these buffers with CUDA.  It is the caller's responsibility to
    // ensure that the blocks meet memory alignment requirements, etc.
    for(i=0; i < ctx->Nb; i++) {
      cuda_rc = hipHostRegister(&ctx->h_blkbufs[i],
                         ctx->Ntpb*ctx->Np*ctx->Nc*sizeof(char2),
                         hipHostRegisterDefault);
      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        return 1;
      }
    }
  }

  // Calculate Ns and allocate host power output buffers
  for(i=0; i < ctx->No; i++) {
    // Ns[i] is number of specta (FFTs) per coarse channel for one input buffer
    // for Nt[i] points per spectra.
    gpu_ctx->Nss[i] = (ctx->Nb * ctx->Ntpb) / ctx->Nts[i];

    // Calculate number of spectra per dump
    gpu_ctx->Nds[i] = gpu_ctx->Nss[i] / ctx->Nas[i];
    if(gpu_ctx->Nds[i] == 0) {
      gpu_ctx->Nds[i] = 1;
    }

    // Calculate number of input buffers per dump
    gpu_ctx->Nis[i] = ctx->Nas[i] / gpu_ctx->Nss[i];
    if(gpu_ctx->Nis[i] == 0) {
      gpu_ctx->Nis[i] = 1;
    }

    // Calculate grid dimensions
    gpu_ctx->grid[i].x = (ctx->Nts[i] + MAX_THREADS - 1) / MAX_THREADS;
    gpu_ctx->grid[i].y = gpu_ctx->Nds[i];
    gpu_ctx->grid[i].z = ctx->Nc;

    // Calculate number of threads per block
    gpu_ctx->nthreads[i] = ctx->Nts[i] < MAX_THREADS ? ctx->Nts[i]
                                                     : MAX_THREADS;

    // Host buffer needs to accommodate the number of integrations that will be
    // dumped at one time (Nd).
    ctx->h_pwrbuf_size[i] = gpu_ctx->Nds[i]*ctx->Nts[i]*ctx->Nc*sizeof(float);
    cuda_rc = hipHostAlloc(&ctx->h_pwrbuf[i], ctx->h_pwrbuf_size[i],
                       hipHostMallocDefault);

    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
  }

  // Allocate buffers

  // FFT input buffer
  // The input buffer is padded to the next multiple of 32KB to facilitate 2D
  // texture lookups by treating the input buffer as a 2D array that is 32KB
  // wide.
  inbuf_size = ctx->Nb*ctx->Ntpb*ctx->Np*ctx->Nc*sizeof(char2);
  if((inbuf_size & 0x7fff) != 0) {
    // Round up to next multiple of 32KB
    inbuf_size = (inbuf_size & ~0x7fff) + 0x8000;
  }

  cuda_rc = hipMalloc(&gpu_ctx->d_fft_in, inbuf_size);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  // Bind texture to device input buffer
  // Width is 32KB, height is inbuf_size/32KB, pitch is 32KB
  cuda_rc = hipBindTexture2D(NULL, char_tex, gpu_ctx->d_fft_in,
                              1<<15, inbuf_size>>15, 1<<15);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  // For each output product
  for(i=0; i < ctx->No; i++) {
    // FFT output buffer
    cuda_rc = hipMalloc(&gpu_ctx->d_fft_out[i], ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(hipfftComplex));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
    // Power output buffer
    cuda_rc = hipMalloc(&gpu_ctx->d_pwr_out[i], ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
    // Clear power output buffer
    cuda_rc = hipMemset(gpu_ctx->d_pwr_out[i], 0, ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
  }

  // Get host pointers to cufft callbacks
  cuda_rc = hipMemcpyFromSymbol(&h_cufft_load_callback,
                                 HIP_SYMBOL(d_cufft_load_callback),
                                 sizeof(h_cufft_load_callback));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  cuda_rc = hipMemcpyFromSymbol(&h_cufft_store_callback,
                                 HIP_SYMBOL(d_cufft_store_callback),
                                 sizeof(h_cufft_store_callback));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  // Generate FFT plans and associate callbacks
  for(i=0; i < ctx->No; i++) {
    // Make the plan
    cufft_rc = hipfftPlanMany(&gpu_ctx->plan[i],      // *plan handle
                             1,                      // rank
                             (int *)&ctx->Nts[i],    // *n
                             (int *)&ctx->Nts[i],    // *inembed (unused for 1d)
                             ctx->Np,                // istride
                             ctx->Nts[i]*ctx->Np,    // idist
                             (int *)&ctx->Nts[i],    // *onembed (unused for 1d)
                             1,                      // ostride
                             ctx->Nts[i],            // odist
                             HIPFFT_C2C,              // type
                             gpu_ctx->Nss[i]*ctx->Nc // batch
                            );

    if(cufft_rc != HIPFFT_SUCCESS) {
      PRINT_ERRMSG(cufft_rc);
      rawspec_cleanup(ctx);
      return 1;
    }

    // Now associate the callbacks with the plan.
    cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i],
                                  (void **)&h_cufft_load_callback,
                                  HIPFFT_CB_LD_COMPLEX,
                                  (void **)&gpu_ctx->d_fft_in);
    if(cufft_rc != HIPFFT_SUCCESS) {
      PRINT_ERRMSG(cufft_rc);
      rawspec_cleanup(ctx);
      return 1;
    }

    cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i],
                                  (void **)&h_cufft_store_callback,
                                  HIPFFT_CB_ST_COMPLEX,
                                  (void **)&gpu_ctx->d_pwr_out[i]);
    if(cufft_rc != HIPFFT_SUCCESS) {
      PRINT_ERRMSG(cufft_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
  }

  // Create streams and associate with plans
  for(i=0; i < ctx->No; i++) {
    cuda_rc = hipStreamCreateWithFlags(&gpu_ctx->stream[i], hipStreamNonBlocking);
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }

    cufft_rc = hipfftSetStream(gpu_ctx->plan[i], gpu_ctx->stream[i]);
    if(cufft_rc != HIPFFT_SUCCESS) {
      PRINT_ERRMSG(cufft_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
  }

  return 0;
}

// Frees host and device buffers based on the ctx->N values.
// Frees and sets the ctx->rawspec_gpu_ctx field.
// Destroys CuFFT plans.
// Destroys streams.
void rawspec_cleanup(rawspec_context * ctx)
{
  int i;
  rawspec_gpu_context * gpu_ctx;

  for(i=0; i<MAX_OUTPUTS; i++) {
    if(ctx->h_pwrbuf[i]) {
      hipHostFree(ctx->h_pwrbuf[i]);
      ctx->h_pwrbuf[i] = NULL;
    }
  }

  if(ctx->gpu_ctx) {
    gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

    if(gpu_ctx->caller_managed) {
      for(i=0; i < ctx->Nb; i++) {
        hipHostUnregister(ctx->h_blkbufs[i]);
      }
    } else {
      if(ctx->h_blkbufs) {
        for(i=0; i < ctx->Nb; i++) {
          hipHostFree(ctx->h_blkbufs[i]);
        }
        free(ctx->h_blkbufs);
        ctx->h_blkbufs = NULL;
      }
    }

    if(gpu_ctx->d_fft_in) {
      hipFree(gpu_ctx->d_fft_in);
    }

    for(i=0; i<MAX_OUTPUTS; i++) {
      if(gpu_ctx->d_fft_out[i]) {
        hipFree(gpu_ctx->d_fft_out[i]);
      }
      if(gpu_ctx->d_pwr_out[i]) {
        hipFree(gpu_ctx->d_pwr_out[i]);
      }
      if(gpu_ctx->plan[i] != NO_PLAN) {
        hipfftDestroy(gpu_ctx->plan[i]);
      }
      if(gpu_ctx->stream[i] != NO_STREAM) {
        hipStreamDestroy(gpu_ctx->stream[i]);
      }
    }

    free(ctx->gpu_ctx);
    ctx->gpu_ctx = NULL;
  }
}

// Copy `ctx->h_blkbufs` to GPU input buffer.
// Returns 0 on success, non-zero on error.
int rawspec_copy_blocks_to_gpu(rawspec_context * ctx,
    off_t src_idx, off_t dst_idx, size_t num_blocks)
{
  int b;
  off_t sblk;
  off_t dblk;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  // TODO Store in GPU context?
  size_t width = ctx->Ntpb * ctx->Np * sizeof(char2);

  for(b=0; b < num_blocks; b++) {
    sblk = (src_idx + b) % ctx->Nb;
    dblk = (dst_idx + b) % ctx->Nb;

    rc = hipMemcpy2D(gpu_ctx->d_fft_in + dblk * width / sizeof(char2),
                      ctx->Nb * width,      // dpitch
                      ctx->h_blkbufs[sblk], // *src
                      width,                // spitch
                      width,                // width
                      ctx->Nc,              // height
                      hipMemcpyHostToDevice);

    if(rc != hipSuccess) {
      PRINT_ERRMSG(rc);
      return 1;
    }
  }

  return 0;
}

// Launches FFTs of data in input buffer.  Whenever an output product
// integration is complete, the power spectrum is copied to the host power
// output buffer and the user provided callback, if any, is called.  This
// function returns zero on success or non-zero if an error is encountered.
//
// The direction of the FFT is determined by the fft_dir parameter.  If fft_dir
// is less than or equal to zero, an inverse (aka backward) transform is
// performed, otherwise a forward transform is performed.
//
// Processing occurs asynchronously.  Use `rawspec_check_for_completion` to
// see how many output products have completed or
// `rawspec_wait_for_completion` to wait for all output products to be
// complete.  New data should NOT be copied to the GPU until
// `rawspec_check_for_completion` returns `ctx->No` or
// `rawspec_wait_for_completion` returns 0.
int rawspec_start_processing(rawspec_context * ctx, int fft_dir)
{
  int i;
  int p;
  int d;
  float * dst;
  size_t dpitch;
  float * src;
  size_t spitch;
  size_t width;
  size_t height;
  hipfftHandle plan;
  hipStream_t stream;
  hipError_t cuda_rc;
  hipfftResult cufft_rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  // Increment inbuf_count
  gpu_ctx->inbuf_count++;

  // For each output product
  for(i=0; i < ctx->No; i++) {

    // Get plan and stream
    plan   = gpu_ctx->plan[i];
    stream = gpu_ctx->stream[i];

    // For each polarization
    for(p=0; p < ctx->Np; p++) {
      // Add FFT to stream
      cufft_rc = hipfftExecC2C(plan,
                              ((hipfftComplex *)gpu_ctx->d_fft_in) + p,
                              gpu_ctx->d_fft_out[i],
                              fft_dir <= 0 ? HIPFFT_BACKWARD : HIPFFT_FORWARD);

      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        return 1;
      }
    }

    // If time to dump
    if(gpu_ctx->inbuf_count % gpu_ctx->Nis[i] == 0) {
      // If the number of spectra to dump per input buffer is less than the
      // number of spectra per input buffer, then we need to accumulate the
      // sub-integrations together.
      if(gpu_ctx->Nds[i] < gpu_ctx->Nss[i]) {
        accumulate<<<gpu_ctx->grid[i],
                     gpu_ctx->nthreads[i],
                     0, stream>>>(gpu_ctx->d_pwr_out[i],
                                  ctx->Nas[i],
                                  ctx->Nts[i],
                                  ctx->Nas[i]*ctx->Nts[i],
                                  ctx->Nb*ctx->Ntpb);
      }

      // Copy integrated power spectra (or spectrum) to host.  This is done as
      // two 2D copies to get channel 0 in the center of the spectrum.  Special
      // care is taken in the unlikely event that Nt is odd.
      src    = gpu_ctx->d_pwr_out[i];
      dst    = ctx->h_pwrbuf[i];
      spitch = gpu_ctx->Nss[i] * ctx->Nts[i] * sizeof(float);
      dpitch = ctx->Nts[i] * sizeof(float);
      height = ctx->Nc;

      for(d=0; d<gpu_ctx->Nds[i]; d++) {

        // Lo to hi
        width  = ((ctx->Nts[i]+1) / 2) * sizeof(float);
        cuda_rc = hipMemcpy2DAsync(dst + ctx->Nts[i]/2,
                                    dpitch,
                                    src,
                                    spitch,
                                    width,
                                    height,
                                    hipMemcpyDeviceToHost,
                                    stream);

        if(cuda_rc != hipSuccess) {
          PRINT_ERRMSG(cuda_rc);
          rawspec_cleanup(ctx);
          return 1;
        }

        // Hi to lo
        width  = (ctx->Nts[i] / 2) * sizeof(float);
        cuda_rc = hipMemcpy2DAsync(dst,
                                    dpitch,
                                    src + (ctx->Nts[i]+1) / 2,
                                    spitch,
                                    width,
                                    height,
                                    hipMemcpyDeviceToHost,
                                    stream);

        if(cuda_rc != hipSuccess) {
          PRINT_ERRMSG(cuda_rc);
          rawspec_cleanup(ctx);
          return 1;
        }

        // Increment src and dst pointers
        src += ctx->Nts[i] * ctx->Nas[i];
        dst += ctx->Nts[i] * ctx->Nc;
      }

      // Add stream callback
      cuda_rc = hipStreamAddCallback(stream, dump_stream_callback,
                                      (void *)&gpu_ctx->dump_cb_data[i], 0);

      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        return 1;
      }

      // Add power buffer clearing hipMemset call to stream
      cuda_rc = hipMemsetAsync(gpu_ctx->d_pwr_out[i], 0,
                                gpu_ctx->Nds[i]*ctx->Nts[i]*ctx->Nc*sizeof(float),
                                stream);

      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        return 1;
      }

    } // If time to dump
  } // For each output product

  return 0;
}

// Waits for any processing to finish, then clears output power buffers and
// resets inbuf_count to 0.  Returns 0 on success, non-zero on error.
int rawspec_reset_integration(rawspec_context * ctx)
{
  int i;
  hipError_t cuda_rc;
  rawspec_gpu_context * gpu_ctx;

  // Mae sure gpu_ctx exists
  if(!ctx->gpu_ctx) {
    return 1;
  }
  gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  // Wait for any/all pending work to complete
  rawspec_wait_for_completion(ctx);

  // For each output product
  for(i=0; i < ctx->No; i++) {
    // Clear power output buffer
    cuda_rc = hipMemset(gpu_ctx->d_pwr_out[i], 0, ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      return 0;
    }
  }

  // Reset inbuf_count
  gpu_ctx->inbuf_count = 0;

  return 0;
}

// Returns the number of output products that are complete for the current
// input buffer.  More precisely, it returns the number of output products that
// are no longer processing (or never were processing) the input buffer.
unsigned int rawspec_check_for_completion(rawspec_context * ctx)
{
  int i;
  int num_complete = 0;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  for(i=0; i<ctx->No; i++) {
    rc = hipStreamQuery(gpu_ctx->stream[i]);
    if(rc == hipSuccess) {
      num_complete++;
    }
  }

  return num_complete;
}

// Waits for any pending output products to be compete processing the current
// input buffer.  Returns zero when complete, non-zero on error.
int rawspec_wait_for_completion(rawspec_context * ctx)
{
  int i;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  for(i=0; i < ctx->No; i++) {
    rc = hipStreamSynchronize(gpu_ctx->stream[i]);
    if(rc != hipSuccess) {
      return 1;
    }
  }

  return 0;
}
