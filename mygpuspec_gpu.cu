#include "hip/hip_runtime.h"
#include "mygpuspec.h"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>

#define NO_PLAN ((hipfftHandle)-1)

#define PRINT_ERRMSG(error)                  \
  fprintf(stderr, "got error %s at %s:%d\n", \
      _cudaGetErrorEnum(error),  \
      __FILE__, __LINE__)

// CPU context structure
typedef struct {
  char2 * d_fft_in; // Device pointer to FFT input buffer
  hipfftComplex * d_fft_out[4]; // Array of device pointers to FFT output buffers
  float * d_pwr_out[4]; // Array of device pointers to power buffers
  hipfftHandle plan[4]; // Array of handles to FFT plans
} mygpuspec_gpu_context;

// Texture declarations
texture<char, 2, hipReadModeNormalizedFloat> char_tex;

__device__ hipfftComplex load_callback(void *p_v_in,
                                      size_t offset,
                                      void *p_v_user,
                                      void *p_v_shared)
{
  hipfftComplex c;
  offset += (hipfftComplex *)p_v_in - (hipfftComplex *)p_v_user;
  c.x = tex2D(char_tex, ((2*offset  ) & 0x7fff), ((  offset  ) >> 14));
  c.y = tex2D(char_tex, ((2*offset+1) & 0x7fff), ((2*offset+1) >> 15));
  return c;
}

__device__ void store_callback(void *p_v_out,
                               size_t offset,
                               hipfftComplex element,
                               void *p_v_user,
                               void *p_v_shared)
{
  float pwr = element.x * element.x + element.y * element.y;
  ((float *)p_v_user)[offset] += pwr;
}

__device__ hipfftCallbackLoadC d_cufft_load_callback = load_callback;
__device__ hipfftCallbackStoreC d_cufft_store_callback = store_callback;

// Sets ctx->Ntmax.
// Allocates host and device buffers based on the ctx->N values.
// Allocates and sets the ctx->mygpuspec_gpu_ctx field.
// Creates CuFFT plans.
// Returns 0 on success, non-zero on error.
int mygpuspec_initialize(mygpuspec_context * ctx)
{
  int i;
  size_t inbuf_size;
  hipError_t cuda_rc;
  hipfftResult cufft_rc;

  // Host copies of cufft callback pointers
  hipfftCallbackLoadC h_cufft_load_callback;
  hipfftCallbackStoreC h_cufft_store_callback;

  // Validate ctx->No
  if(ctx->No == 0 || ctx->No > MAX_OUTPUTS) {
    fprintf(stderr, "output products must be in range [1..%d], not %d\n",
        MAX_OUTPUTS, ctx->No);
    return 1;
  }

  // Validate Ntpb
  if(ctx->Ntpb == 0) {
    fprintf(stderr, "number of time samples per block cannot be zero\n");
    return 1;
  }


  // Determine Ntmax (and validate Nts)
  ctx->Ntmax = 0;
  for(i=0; i<ctx->No; i++) {
    if(ctx->Nts[i] == 0) {
      fprintf(stderr, "Nts[%d] cannot be 0\n", i);
      return 1;
    }
    if(ctx->Ntmax < ctx->Nts[i]) {
      ctx->Ntmax = ctx->Nts[i];
    }
  }
  // Validate that all Nts are factors of Ntmax.  This constraint helps
  // simplify input buffer management.
  for(i=0; i<ctx->No; i++) {
    if(ctx->Ntmax % ctx->Nts[i] != 0) {
      fprintf(stderr, "Nts[%d] (%u) is not a factor of Ntmax (%u)\n",
          i, ctx->Nts[i], ctx->Ntmax);
      return 1;
    }
  }

  // Validate/calculate Nb
  // If ctx->Nb is given by caller (i.e. is non-zero)
  if(ctx->Nb != 0) {
    // Validate that Ntmax is a factor of (Nb * Ntpb)
    if((ctx->Nb * ctx->Ntpb) % ctx->Ntmax != 0) {
      fprintf(stderr,
          "Ntmax (%u) is not a factor of Nb*Ntpb (%u * %u = %u)\n",
          ctx->Ntmax, ctx->Nb, ctx->Ntpb, ctx->Nb*ctx->Ntpb);
      return 1;
    }
  } else {
    // Calculate Nb
    // If Ntmax is less than one block
    if(ctx->Ntmax < ctx->Ntpb) {
      // Validate that Ntmax is a factor of Ntpb
      if(ctx->Ntpb % ctx->Ntmax != 0) {
        fprintf(stderr, "Ntmax (%u) is not a factor of Ntpb (%u)\n",
            ctx->Ntmax, ctx->Ntpb);
        return 1;
      }
      ctx->Nb = 1;
    } else {
      // Validate that Ntpb is factor of Ntmax
      if(ctx->Ntmax % ctx->Ntpb != 0) {
        fprintf(stderr, "Ntpb (%u) is not a factor of Nmax (%u)\n",
            ctx->Ntpb, ctx->Ntmax);
        return 1;
      }
      ctx->Nb = ctx->Ntmax / ctx->Ntpb;
    }
  }

  // Null out all pointers
  ctx->h_blkbufs = NULL;
  for(i=0; i < MAX_OUTPUTS; i++) {
    ctx->h_pwrbuf[i] = NULL;
  }
  ctx->gpu_ctx = NULL;

  // Alllocate host buffers
  ctx->h_blkbufs = (char **)malloc(ctx->Nb * sizeof(char *));
  for(i=0; i < ctx->Nb; i++) {
    // Block buffer can use write combining
    cuda_rc = hipHostAlloc(&ctx->h_blkbufs[i],
                       ctx->Ntpb*ctx->Np*ctx->Nc*sizeof(char2),
                       hipHostMallocWriteCombined);
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      return 1;
    }
  }

  for(i=0; i < ctx->No; i++) {
    // TODO For small Nt values, it's probbaly more efficient to buffer
    // multiple power spectra in the output buffer, but this requires a little
    // more overhead so it is deferred for now.
    cuda_rc = hipHostAlloc(&ctx->h_pwrbuf[i],
                       ctx->Nts[i]*ctx->Nc*sizeof(float),
                       hipHostMallocDefault);
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      mygpuspec_cleanup(ctx);
      return 1;
    }
  }

  // Allocate GPU context
  mygpuspec_gpu_context * gpu_ctx = (mygpuspec_gpu_context *)malloc(sizeof(mygpuspec_gpu_context));

  if(!gpu_ctx) {
    mygpuspec_cleanup(ctx);
    return 1;
  }

  // Store pointer to gpu_ctx in ctx
  ctx->gpu_ctx = gpu_ctx;

  // NULL out pointers (and invalidate plans)
  gpu_ctx->d_fft_in = NULL;
  for(i=0; i<MAX_OUTPUTS; i++) {
    gpu_ctx->d_fft_out[i] = NULL;
    gpu_ctx->d_pwr_out[i] = NULL;
    gpu_ctx->plan[i] = NO_PLAN;
  }

  // Allocate buffers

  // FFT input buffer
  // The input buffer is padded to the next multiple of 32KB to facilitate 2D
  // texture lookups by treating the input buffer as a 2D array that is 32KB
  // wide.
  inbuf_size = ctx->Ntmax*ctx->Np*ctx->Nc*sizeof(char2);
  if((inbuf_size & 0x7fff) != 0) {
    // Round up to next multiple of 32KB
    inbuf_size = (inbuf_size & ~0x7fff) + 0x8000;
  }

  cuda_rc = hipMalloc(&gpu_ctx->d_fft_in, inbuf_size);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    mygpuspec_cleanup(ctx);
    return 1;
  }

  // Bind texture to device input buffer
  // Width is 32KB, height is inbuf_size/32KB, pitch is 32KB
  cuda_rc = hipBindTexture2D(NULL, char_tex, gpu_ctx->d_fft_in,
                              1<<15, inbuf_size>>15, 1<<15);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    mygpuspec_cleanup(ctx);
    return 1;
  }

  // For each output product
  for(i=0; i < ctx->No; i++) {
    // FFT output buffer
    cuda_rc = hipMalloc(&gpu_ctx->d_fft_out[i], ctx->Nts[i]*ctx->Nc*sizeof(hipfftComplex));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      mygpuspec_cleanup(ctx);
      return 1;
    }
    // Power output buffer
    cuda_rc = hipMalloc(&gpu_ctx->d_pwr_out[i], ctx->Nts[i]*ctx->Nc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      mygpuspec_cleanup(ctx);
      return 1;
    }
    // Clear power output buffer
    cuda_rc = hipMemset(gpu_ctx->d_pwr_out[i], 0, ctx->Nts[i]*ctx->Nc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      mygpuspec_cleanup(ctx);
      return 1;
    }
  }

  // Get host pointers to cufft callbacks
  cuda_rc = hipMemcpyFromSymbol(&h_cufft_load_callback,
                                 HIP_SYMBOL(d_cufft_load_callback),
                                 sizeof(h_cufft_load_callback));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    mygpuspec_cleanup(ctx);
    return 1;
  }

  cuda_rc = hipMemcpyFromSymbol(&h_cufft_store_callback,
                                 HIP_SYMBOL(d_cufft_store_callback),
                                 sizeof(h_cufft_store_callback));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    mygpuspec_cleanup(ctx);
    return 1;
  }

  // Generate FFT plans and associate callbacks
  for(i=0; i < ctx->No; i++) {
    // Make the plan
    cufft_rc = hipfftPlanMany(&gpu_ctx->plan[i],   // *plan handle
                             1,                   // rank
                             (int *)&ctx->Nts[i], // *n
                             (int *)&ctx->Nts[i], // *inembed (unused for 1d)
                             ctx->Np,             // istride
                             ctx->Nts[i]*ctx->Np, // idist
                             (int *)&ctx->Nts[i], // *onembed (unused for 1d)
                             1,                   // ostride
                             ctx->Nts[i],         // odist
                             HIPFFT_C2C,           // type
                             ctx->Nc              // batch
                            );

    if(cufft_rc != HIPFFT_SUCCESS) {
      PRINT_ERRMSG(cufft_rc);
      mygpuspec_cleanup(ctx);
      return 1;
    }

    // Now associate the callbacks with the plan.
    cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i],
                                  (void **)&h_cufft_load_callback,
                                  HIPFFT_CB_LD_COMPLEX,
                                  (void **)&gpu_ctx->d_fft_in);
    if(cufft_rc != HIPFFT_SUCCESS) {
      PRINT_ERRMSG(cufft_rc);
      mygpuspec_cleanup(ctx);
      return 1;
    }

    cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i],
                                  (void **)&h_cufft_store_callback,
                                  HIPFFT_CB_ST_COMPLEX,
                                  (void **)&gpu_ctx->d_pwr_out[i]);
    if(cufft_rc != HIPFFT_SUCCESS) {
      PRINT_ERRMSG(cufft_rc);
      mygpuspec_cleanup(ctx);
      return 1;
    }
  }

  return 0;
}

// Frees host and device buffers based on the ctx->N values.
// Frees and sets the ctx->mygpuspec_gpu_ctx field.
// Destroys CuFFT plans.
void mygpuspec_cleanup(mygpuspec_context * ctx)
{
  int i;
  mygpuspec_gpu_context * gpu_ctx;

  if(ctx->h_blkbufs) {
    for(i=0; i < ctx->Nb; i++) {
      hipHostFree(ctx->h_blkbufs[i]);
    }
    free(ctx->h_blkbufs);
    ctx->h_blkbufs = NULL;
  }

  for(i=0; i<MAX_OUTPUTS; i++) {
    if(ctx->h_pwrbuf[i]) {
      hipHostFree(ctx->h_pwrbuf[i]);
      ctx->h_pwrbuf[i] = NULL;
    }
  }

  if(ctx->gpu_ctx) {
    gpu_ctx = (mygpuspec_gpu_context *)ctx->gpu_ctx;

    if(gpu_ctx->d_fft_in) {
      hipFree(gpu_ctx->d_fft_in);
    }

    for(i=0; i<MAX_OUTPUTS; i++) {
      if(gpu_ctx->d_fft_out[i]) {
        hipFree(gpu_ctx->d_fft_out[i]);
      }
      if(gpu_ctx->d_pwr_out[i]) {
        hipFree(gpu_ctx->d_pwr_out[i]);
      }
      if(gpu_ctx->plan[i] != NO_PLAN) {
        hipfftDestroy(gpu_ctx->plan[i]);
      }
    }

    free(ctx->gpu_ctx);
    ctx->gpu_ctx = NULL;
  }
}

// Copy `ctx->h_blkbufs` to GPU input buffer.
// Returns 0 on success, non-zero on error.
int mygpuspec_copy_blocks_to_gpu(mygpuspec_context * ctx)
{
  int b;
  hipError_t rc;
  mygpuspec_gpu_context * gpu_ctx = (mygpuspec_gpu_context *)ctx->gpu_ctx;

  // TODO Store in GPU context?
  size_t width = ctx->Ntpb * ctx->Np * sizeof(char2);

  for(b=0; b < ctx->Nb; b++) {
    rc = hipMemcpy2D(gpu_ctx->d_fft_in + b * width / sizeof(char2),
                      ctx->Nb * width,   // dpitch
                      ctx->h_blkbufs[b], // *src
                      width,             // spitch
                      width,             // width
                      ctx->Nc,           // height
                      hipMemcpyHostToDevice);

    if(rc != hipSuccess) {
      PRINT_ERRMSG(rc);
      return 1;
    }
  }

  return 0;
}
