#include "hip/hip_runtime.h"
#include "mygpuspec_gpu.h"

#if 0
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
//#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Texture declarations
texture<char, 1, hipReadModeNormalizedFloat> char_tex;

__device__ hipfftComplex load_callback(void *p_v_in, 
                                      size_t offset, 
                                      void *p_v_user,
                                      void *p_v_shared)
{
  hipfftComplex c;
  offset += (hipfftComplex *)p_v_in - (hipfftComplex *)p_v_user;
  c.x = tex1Dfetch(char_tex, 2*offset  );
  c.y = tex1Dfetch(char_tex, 2*offset+1);
  return c;
}

__device__ void store_callback(void *p_v_out, 
                               size_t offset, 
                               hipfftComplex element,
                               void *p_v_user,
                               void *p_v_shared)
{
  float pwr = element.x * element.x + element.y * element.y;
  ((float *)p_v_user)[offset] += pwr;
}

__device__ hipfftCallbackLoadC d_pcb_load_callback = load_callback;
__device__ hipfftCallbackStoreC d_pcb_store_callback = store_callback;

int runTest(int argc, char **argv)
{
  int i, j;

  // Pointers to host memory buffers
  char2 * h_pc2_in;
  float * h_pf_out;

  // Pointers to device memory buffers
  hipfftComplex * d_pc2_in;  // FFT input buffer  (char2 really)
  hipfftComplex * d_pf2_out; // FFT output buffer (must be full sized and can't integrate in it)
  float        * d_pf_out;  // Power output buffer (integrate power here)

  // FFT plan related variables
  const int Nt   = 512; // FFT size
  const int Nb   = 3;   // batch size 
  const int Ni   = 2;   // interleave size
  const int Nti  = Nt * Ni;
  const int Ntb  = Nt * Nb;
  const int Ntbi = Nt * Nb * Ni;
  int Nt_ = Nt; // So we can make a non-const (int *) from it
  hipfftHandle plan;
  size_t work_size = 0;

  // Host copies of callback pointers
  hipfftCallbackLoadC h_pcb_load_callback;
  hipfftCallbackStoreC h_pcb_store_callback;

  // Allocate host memory
  h_pc2_in  = (char2 *)malloc(Ntbi*sizeof(char2));
  h_pf_out  = (float *)malloc(Ntb *sizeof(float));
  if(!h_pc2_in || !h_pf_out) {
    fprintf(stderr, "could not allocate host memory\n");
    return 1;
  }

  // Allocate device memory
  checkCudaErrors(hipMalloc((void **)&d_pc2_in,  Ntbi*sizeof(char2)));
  checkCudaErrors(hipMalloc((void **)&d_pf2_out, Ntb *sizeof(float2)));
  checkCudaErrors(hipMalloc((void **)&d_pf_out,  Ntb *sizeof(float)));
  // Clear power output buffer
  checkCudaErrors(hipMemset(d_pf_out, 0, Ntb*sizeof(float)));

  // Bind texture to device input buffer
  checkCudaErrors(hipBindTexture(NULL, char_tex, d_pc2_in, Ntbi*sizeof(char2)));

  // Allocate plan memory
  checkCudaErrors(hipfftCreate(&plan));
  // Make the plan
  checkCudaErrors(hipfftMakePlanMany(plan,      // plan handle
                                    1,         // rank
                                    &Nt_,      // *n
                                    &Nt_,      // *inembed (unused for 1d)
                                    Ni,        // istride
                                    Nti,       // idist
                                    &Nt_,      // *onembed (unused for 1d)
                                    1,         // ostride
                                    Nt,        // odist
                                    HIPFFT_C2C, // type
                                    Nb,        // batch
                                    &work_size // worksize
                                   ));

  printf("Temporary buffer size %li bytes\n", work_size);

  // Setup the callbacks
  hipMemcpyFromSymbol(&h_pcb_load_callback, 
                       HIP_SYMBOL(d_pcb_load_callback), 
                       sizeof(h_pcb_load_callback));

  hipMemcpyFromSymbol(&h_pcb_store_callback, 
                       HIP_SYMBOL(d_pcb_store_callback), 
                       sizeof(h_pcb_store_callback));

  // Now associate the callbacks with the plan.
  hipfftResult status = hipfftXtSetCallback(plan,
                                          (void **)&h_pcb_load_callback,
                                          HIPFFT_CB_LD_COMPLEX,
                                          (void **)&d_pc2_in);
  if (status == CUFFT_LICENSE_ERROR)
  {
      printf("Apparently, using CUFFT callbacks requires a valid license file.\n");
      printf("The file was either not found, out of date, or otherwise invalid.\n");
      return 1;
  }
  checkCudaErrors(hipfftXtSetCallback(plan,
                                     (void **)&h_pcb_load_callback,
                                     HIPFFT_CB_LD_COMPLEX,
                                     (void **)&d_pc2_in));

  checkCudaErrors(hipfftXtSetCallback(plan,
                                     (void **)&h_pcb_store_callback,
                                     HIPFFT_CB_ST_COMPLEX,
                                     (void **)&d_pf_out));

  // Populate input data
  memset(h_pc2_in, 0, Ntbi * sizeof(char2));
  for(i=0; i<Nt; i++) {
    // Even samples, odd bins
    h_pc2_in[2*(i       )].x = round(127*cos(2*M_PI*1*i/Nt)); // Bin 1
    h_pc2_in[2*(i       )].y = round(127*sin(2*M_PI*1*i/Nt)); // Bin 1
    h_pc2_in[2*(i +   Nt)].x = round(127*cos(2*M_PI*3*i/Nt)); // Bin 3
    h_pc2_in[2*(i +   Nt)].y = round(127*sin(2*M_PI*3*i/Nt)); // Bin 3
    h_pc2_in[2*(i + 2*Nt)].x = round(127*cos(2*M_PI*5*i/Nt)); // Bin 5;
    h_pc2_in[2*(i + 2*Nt)].y = round(127*sin(2*M_PI*5*i/Nt)); // Bin 5;

    // Odd samples, even bind, half power relative to even samples
    h_pc2_in[2*(i       )+1].x = round(89.8*cos(2*M_PI*2*i/Nt)); // Bin 2
    h_pc2_in[2*(i       )+1].y = round(89.8*sin(2*M_PI*2*i/Nt)); // Bin 2
    h_pc2_in[2*(i +   Nt)+1].x = round(89.8*cos(2*M_PI*4*i/Nt)); // Bin 4
    h_pc2_in[2*(i +   Nt)+1].y = round(89.8*sin(2*M_PI*4*i/Nt)); // Bin 4
    h_pc2_in[2*(i + 2*Nt)+1].x = round(89.8*cos(2*M_PI*6*i/Nt)); // Bin 6;
    h_pc2_in[2*(i + 2*Nt)+1].y = round(89.8*sin(2*M_PI*6*i/Nt)); // Bin 6;
  }

  // Copy data to GPU
  checkCudaErrors(hipMemcpy(d_pc2_in, h_pc2_in, Ntbi*sizeof(char2),
                             hipMemcpyHostToDevice));

  for(j=0; j<Ni; j++) {
    // Do FFT, integrating all interleaved outputs together
    checkCudaErrors(hipfftExecC2C(plan, d_pc2_in+j, d_pf2_out, HIPFFT_FORWARD));
  }

  // Copy data back from the GPU
  checkCudaErrors(hipMemcpy(h_pf_out, d_pf_out, Ntb*sizeof(float),
                             hipMemcpyDeviceToHost));

  // Show output
  for(i=0; i<17; i++) {
    printf("%3d", i);
    printf("  %+11.8f",   h_pf_out[i     ]/(512*512));
    printf("  %+11.8f",   h_pf_out[i+  Nt]/(512*512));
    printf("  %+11.8f\n", h_pf_out[i+2*Nt]/(512*512));
  }
  printf("\n");

  //Destroy CUFFT plan
  checkCudaErrors(hipfftDestroy(plan));

  // Ub-bind texture from device input buffer
  hipUnbindTexture(char_tex);

  // Free device memory
  checkCudaErrors(hipFree(d_pf2_out));
  checkCudaErrors(hipFree(d_pc2_in));
  free(h_pf_out);
  free(h_pc2_in);

  return 0;
}
#endif // 0
